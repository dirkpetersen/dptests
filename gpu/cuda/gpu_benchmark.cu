#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <thread>
#include <iomanip>
#include <cmath>

#define CHECK_CUDA(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(error) << std::endl; \
            exit(1); \
        } \
    } while(0)

#define CHECK_CUBLAS(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(1); \
        } \
    } while(0)

__global__ void memoryBandwidthKernel(float* data, size_t size, int iterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int iter = 0; iter < iterations; iter++) {
        for (size_t i = idx; i < size; i += stride) {
            data[i] = data[i] * 1.001f + 0.001f;
        }
    }
}

__global__ void computeIntensiveKernel(float* data, size_t size, int iterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int iter = 0; iter < iterations; iter++) {
        for (size_t i = idx; i < size; i += stride) {
            float val = data[i];
            val = sinf(val) + cosf(val);
            val = expf(val * 0.1f);
            val = logf(val + 1.0f);
            val = sqrtf(val);
            data[i] = val;
        }
    }
}

class GPUBenchmark {
private:
    int numGPUs;
    std::vector<int> gpuIds;
    std::vector<hipStream_t> streams;
    std::vector<hipblasHandle_t> cublasHandles;
    std::vector<float*> deviceMemory;
    std::vector<size_t> memoryPerGPU;
    
public:
    GPUBenchmark(int num_gpus) : numGPUs(num_gpus) {
        int totalGPUs;
        CHECK_CUDA(hipGetDeviceCount(&totalGPUs));
        
        if (numGPUs > totalGPUs) {
            std::cerr << "Requested " << numGPUs << " GPUs but only " << totalGPUs << " available" << std::endl;
            numGPUs = totalGPUs;
        }
        
        gpuIds.resize(numGPUs);
        streams.resize(numGPUs);
        cublasHandles.resize(numGPUs);
        deviceMemory.resize(numGPUs);
        memoryPerGPU.resize(numGPUs);
        
        for (int i = 0; i < numGPUs; i++) {
            gpuIds[i] = i;
            CHECK_CUDA(hipSetDevice(i));
            
            size_t free, total;
            CHECK_CUDA(hipMemGetInfo(&free, &total));
            memoryPerGPU[i] = free * 0.8; // Use 80% of available memory
            
            CHECK_CUDA(hipMalloc(&deviceMemory[i], memoryPerGPU[i]));
            CHECK_CUDA(hipStreamCreate(&streams[i]));
            CHECK_CUBLAS(hipblasCreate(&cublasHandles[i]));
            CHECK_CUBLAS(hipblasSetStream(cublasHandles[i], streams[i]));
            
            std::cout << "GPU " << i << ": Allocated " << memoryPerGPU[i] / (1024*1024*1024) << " GB" << std::endl;
        }
    }
    
    ~GPUBenchmark() {
        for (int i = 0; i < numGPUs; i++) {
            CHECK_CUDA(hipSetDevice(i));
            if (deviceMemory[i]) CHECK_CUDA(hipFree(deviceMemory[i]));
            if (streams[i]) CHECK_CUDA(hipStreamDestroy(streams[i]));
            if (cublasHandles[i]) CHECK_CUBLAS(hipblasDestroy(cublasHandles[i]));
        }
    }
    
    void initializeData() {
        std::cout << "Initializing data on all GPUs..." << std::endl;
        
        for (int i = 0; i < numGPUs; i++) {
            CHECK_CUDA(hipSetDevice(i));
            
            hiprandGenerator_t gen;
            hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
            hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL) + i);
            hiprandSetStream(gen, streams[i]);
            
            size_t numElements = memoryPerGPU[i] / sizeof(float);
            hiprandGenerateUniform(gen, deviceMemory[i], numElements);
            
            hiprandDestroyGenerator(gen);
        }
        
        for (int i = 0; i < numGPUs; i++) {
            CHECK_CUDA(hipSetDevice(i));
            CHECK_CUDA(hipStreamSynchronize(streams[i]));
        }
    }
    
    void runMemoryBandwidthTest(int durationMs) {
        std::cout << "Running memory bandwidth test..." << std::endl;
        
        auto start = std::chrono::high_resolution_clock::now();
        auto end = start + std::chrono::milliseconds(durationMs);
        
        int iterations = 0;
        while (std::chrono::high_resolution_clock::now() < end) {
            for (int i = 0; i < numGPUs; i++) {
                CHECK_CUDA(hipSetDevice(i));
                
                size_t numElements = memoryPerGPU[i] / sizeof(float);
                int numBlocks = std::min(65535, (int)((numElements + 255) / 256));
                int threadsPerBlock = 256;
                
                memoryBandwidthKernel<<<numBlocks, threadsPerBlock, 0, streams[i]>>>(
                    deviceMemory[i], numElements, 10);
            }
            iterations++;
        }
        
        for (int i = 0; i < numGPUs; i++) {
            CHECK_CUDA(hipSetDevice(i));
            CHECK_CUDA(hipStreamSynchronize(streams[i]));
        }
        
        auto actualEnd = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(actualEnd - start);
        
        size_t totalBytes = 0;
        for (int i = 0; i < numGPUs; i++) {
            totalBytes += memoryPerGPU[i];
        }
        
        double bandwidth = (double)(totalBytes * iterations * 2) / (1024.0 * 1024.0 * 1024.0) / (duration.count() / 1000.0);
        std::cout << "Memory bandwidth: " << std::fixed << std::setprecision(2) << bandwidth << " GB/s" << std::endl;
    }
    
    void runComputeTest(int durationMs) {
        std::cout << "Running compute-intensive test..." << std::endl;
        
        auto start = std::chrono::high_resolution_clock::now();
        auto end = start + std::chrono::milliseconds(durationMs);
        
        int iterations = 0;
        while (std::chrono::high_resolution_clock::now() < end) {
            for (int i = 0; i < numGPUs; i++) {
                CHECK_CUDA(hipSetDevice(i));
                
                size_t numElements = memoryPerGPU[i] / sizeof(float);
                int numBlocks = std::min(65535, (int)((numElements + 255) / 256));
                int threadsPerBlock = 256;
                
                computeIntensiveKernel<<<numBlocks, threadsPerBlock, 0, streams[i]>>>(
                    deviceMemory[i], numElements, 100);
            }
            iterations++;
        }
        
        for (int i = 0; i < numGPUs; i++) {
            CHECK_CUDA(hipSetDevice(i));
            CHECK_CUDA(hipStreamSynchronize(streams[i]));
        }
        
        auto actualEnd = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(actualEnd - start);
        
        size_t totalElements = 0;
        for (int i = 0; i < numGPUs; i++) {
            totalElements += memoryPerGPU[i] / sizeof(float);
        }
        
        double flops = (double)(totalElements * iterations * 100 * 6) / (duration.count() / 1000.0) / 1e12;
        std::cout << "Compute performance: " << std::fixed << std::setprecision(2) << flops << " TFLOPS" << std::endl;
    }
    
    void runMatrixMultiplication(int durationMs) {
        std::cout << "Running matrix multiplication test..." << std::endl;
        
        std::vector<float*> matrixA(numGPUs);
        std::vector<float*> matrixB(numGPUs);
        std::vector<float*> matrixC(numGPUs);
        
        for (int i = 0; i < numGPUs; i++) {
            CHECK_CUDA(hipSetDevice(i));
            
            size_t availableElements = memoryPerGPU[i] / sizeof(float);
            int matrixSize = (int)sqrt(availableElements / 3);
            matrixSize = (matrixSize / 32) * 32; // Align to 32
            
            size_t matrixBytes = matrixSize * matrixSize * sizeof(float);
            
            CHECK_CUDA(hipMalloc(&matrixA[i], matrixBytes));
            CHECK_CUDA(hipMalloc(&matrixB[i], matrixBytes));
            CHECK_CUDA(hipMalloc(&matrixC[i], matrixBytes));
            
            hiprandGenerator_t gen;
            hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
            hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL) + i);
            hiprandSetStream(gen, streams[i]);
            
            hiprandGenerateUniform(gen, matrixA[i], matrixSize * matrixSize);
            hiprandGenerateUniform(gen, matrixB[i], matrixSize * matrixSize);
            
            hiprandDestroyGenerator(gen);
            
            std::cout << "GPU " << i << ": Matrix size " << matrixSize << "x" << matrixSize << std::endl;
        }
        
        auto start = std::chrono::high_resolution_clock::now();
        auto end = start + std::chrono::milliseconds(durationMs);
        
        int iterations = 0;
        while (std::chrono::high_resolution_clock::now() < end) {
            for (int i = 0; i < numGPUs; i++) {
                CHECK_CUDA(hipSetDevice(i));
                
                size_t availableElements = memoryPerGPU[i] / sizeof(float);
                int matrixSize = (int)sqrt(availableElements / 3);
                matrixSize = (matrixSize / 32) * 32;
                
                const float alpha = 1.0f, beta = 0.0f;
                CHECK_CUBLAS(hipblasSgemm(cublasHandles[i],
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    matrixSize, matrixSize, matrixSize,
                    &alpha,
                    matrixA[i], matrixSize,
                    matrixB[i], matrixSize,
                    &beta,
                    matrixC[i], matrixSize));
            }
            iterations++;
        }
        
        for (int i = 0; i < numGPUs; i++) {
            CHECK_CUDA(hipSetDevice(i));
            CHECK_CUDA(hipStreamSynchronize(streams[i]));
        }
        
        auto actualEnd = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(actualEnd - start);
        
        double totalFlops = 0;
        for (int i = 0; i < numGPUs; i++) {
            size_t availableElements = memoryPerGPU[i] / sizeof(float);
            int matrixSize = (int)sqrt(availableElements / 3);
            matrixSize = (matrixSize / 32) * 32;
            totalFlops += 2.0 * matrixSize * matrixSize * matrixSize;
        }
        
        double tflops = totalFlops * iterations / (duration.count() / 1000.0) / 1e12;
        std::cout << "Matrix multiplication: " << std::fixed << std::setprecision(2) << tflops << " TFLOPS" << std::endl;
        
        for (int i = 0; i < numGPUs; i++) {
            CHECK_CUDA(hipSetDevice(i));
            CHECK_CUDA(hipFree(matrixA[i]));
            CHECK_CUDA(hipFree(matrixB[i]));
            CHECK_CUDA(hipFree(matrixC[i]));
        }
    }
    
    void printGPUInfo() {
        std::cout << "\n=== GPU Information ===" << std::endl;
        for (int i = 0; i < numGPUs; i++) {
            CHECK_CUDA(hipSetDevice(i));
            
            hipDeviceProp_t prop;
            CHECK_CUDA(hipGetDeviceProperties(&prop, i));
            
            std::cout << "GPU " << i << ": " << prop.name << std::endl;
            std::cout << "  Compute Capability: " << prop.major << "." << prop.minor << std::endl;
            std::cout << "  Memory: " << prop.totalGlobalMem / (1024*1024*1024) << " GB" << std::endl;
            std::cout << "  SMs: " << prop.multiProcessorCount << std::endl;
            std::cout << "  Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
        }
        std::cout << std::endl;
    }
};

int main(int argc, char* argv[]) {
    int numGPUs = 1;
    
    if (argc > 1) {
        numGPUs = std::atoi(argv[1]);
        if (numGPUs <= 0) {
            std::cerr << "Invalid number of GPUs: " << numGPUs << std::endl;
            return 1;
        }
    }
    
    std::cout << "=== Multi-GPU CUDA Benchmark ===" << std::endl;
    std::cout << "Requested GPUs: " << numGPUs << std::endl;
    
    try {
        GPUBenchmark benchmark(numGPUs);
        benchmark.printGPUInfo();
        benchmark.initializeData();
        
        const int testDuration = 15000; // 15 seconds per test
        
        benchmark.runMemoryBandwidthTest(testDuration);
        benchmark.runComputeTest(testDuration);
        benchmark.runMatrixMultiplication(testDuration);
        
        std::cout << "\n=== Benchmark Complete ===" << std::endl;
        
    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }
    
    return 0;
}